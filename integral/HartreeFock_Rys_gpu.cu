#include "Hamiltonian.cuh"
#include "hcore.cuh"
#include "JKmat.cuh"
#include <filesystem>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <thrust/extrema.h>

using namespace hcore_gpu;


HartreeFock_Rys_gpu::~HartreeFock_Rys_gpu()
{
    if (Schwarz_mat != nullptr)
        hipFree(Schwarz_mat);
    if (rys_root != nullptr)
        hipFree(rys_root);
}

int HartreeFock_Rys_gpu::init()
{
    // initialize the molecule basis on gpu
    if (sort_AO){
        if(m_molbasis.Sort_AOs()){
            std::cout << "sort AO failed!" << std::endl;
            return 1;
        }
        copy_sorted_molecule_basis_to_gpu(m_molbasis, m_molbasis_gpu);
    }else{
        copy_molecule_basis_to_gpu(m_molbasis, m_molbasis_gpu);
    }

    size_t dim = m_molbasis.mAOs.size();

    // checking the basis set to see if there is high angular momentum stuff
    for (int mu = 0; mu < dim; mu++)
    {
        if (arma::accu(m_molbasis.mAOs[mu].lmn) >= 2)
        {
            std::cout << "higher angular momentum basis detected! Can only do s and p";
            return 1;
        }
    }

    // loading rys roots
    std::string aux;
    if (const char *env_p = std::getenv("GPUChem_aux"))
    {
        aux = std::string(env_p);
        if (!std::filesystem::is_directory(aux))
        {
            throw std::runtime_error("basis/basis_set.cpp: The directory specified by GPUChem_aux does not exist!");
        }
    }
    arma::mat rys_root_arma;
    // text file contatins rys root (squared) and their weights from X = 0 to 30 (0.01 increment)
    rys_root_arma.load(aux + "/rys_root.txt");

    // Get the size of rys_root_arma and Schwarz_mat_arma
    Schwarz_mat_dim0 = dim;
    Schwarz_mat_dim1 = dim;
    rys_root_dim0 = rys_root_arma.n_cols;
    rys_root_dim1 = rys_root_arma.n_rows;

    // Allocate memory on GPU for rys_root and Schwarz_mat
    hipMalloc((void **)&rys_root, sizeof(double) * rys_root_arma.n_elem);
    hipMalloc((void **)&Schwarz_mat, sizeof(double) * dim * dim);

    // Copy rys_root from CPU to GPU
    hipMemcpy(rys_root, rys_root_arma.memptr(), sizeof(double) * rys_root_arma.n_elem, hipMemcpyHostToDevice);

    // evaluate the Schwarz matrix
    // set dim3 grid and block to 2D
    dim3 blockDim(8, 8);
    dim3 gridDim((dim + blockDim.x - 1) / blockDim.x, (dim + blockDim.y - 1) / blockDim.y);
    // call the kernel
    eval_Schwarzmat_GPU<<<gridDim, blockDim>>>(m_molbasis_gpu.mAOs, rys_root, Schwarz_mat, dim, rys_root_dim1);




    return 0;
}

int HartreeFock_Rys_gpu::eval_OV(arma::mat &OV_mat)
{
    int ok = eval_OVmat_without_sort_inside(m_molbasis_gpu, OV_mat);
    // int ok =  eval_OVmat(m_molbasis_gpu, m_molbasis, OV_mat);
    if (sort_AO)
        OV_mat = OV_mat(m_molbasis.mAOs_sorted_index_inv, m_molbasis.mAOs_sorted_index_inv);

    return ok;
}

int HartreeFock_Rys_gpu::eval_Hcore(arma::mat &H_mat)
{
    // evaluate the H core matrix (one-electron part)
    int ok = eval_Hcoremat_without_sort_inside(m_molbasis_gpu, H_mat);
    // int ok = eval_Hcoremat(m_molbasis_gpu, m_molbasis, H_mat);
    if (sort_AO)
        H_mat = H_mat(m_molbasis.mAOs_sorted_index_inv, m_molbasis.mAOs_sorted_index_inv);
    
    return ok;
}

int HartreeFock_Rys_gpu::eval_G(arma::mat &P_mat, arma::mat &G_mat)
{
    size_t dim = m_molbasis.mAOs.size();
    // check the size of P_mat and G_mat
    if (P_mat.n_rows != dim || P_mat.n_cols != dim || G_mat.n_rows != dim || G_mat.n_cols != dim)
    {
        std::cout << "P_mat or G_mat has wrong dimension!";
        return 1;
    }
    
    double *P_mat_gpu, *G_mat_gpu;
    hipMalloc((void **)&P_mat_gpu, sizeof(double) * P_mat.n_elem);
    hipMalloc((void **)&G_mat_gpu, sizeof(double) * G_mat.n_elem);
    
    if (sort_AO){
        arma::mat P_mat_temp = P_mat(m_molbasis.mAOs_sorted_index, m_molbasis.mAOs_sorted_index);
        hipMemcpy(P_mat_gpu, P_mat_temp.memptr(), sizeof(double) * P_mat.n_elem, hipMemcpyHostToDevice);
    }else
        hipMemcpy(P_mat_gpu, P_mat.memptr(), sizeof(double) * P_mat.n_elem, hipMemcpyHostToDevice);

    // evaluate the G matrix (two-electron part)
    int ok = 0;
    ok = eval_Gmat_RSCF(m_molbasis_gpu, rys_root, Schwarz_mat, shreshold, P_mat_gpu, G_mat_gpu, rys_root_dim1);

    // copy G_mat from GPU to CPU
    hipMemcpy(G_mat.memptr(), G_mat_gpu, sizeof(double) * G_mat.n_elem, hipMemcpyDeviceToHost);

    if (sort_AO)
        G_mat = G_mat(m_molbasis.mAOs_sorted_index_inv, m_molbasis.mAOs_sorted_index_inv);

    return ok;
}

int HartreeFock_Rys_gpu::eval_J(arma::mat &P_mat, arma::mat &J_mat)
{

    size_t dim = m_molbasis.mAOs.size();
    // check the size of P_mat and J_mat
    if (P_mat.n_rows != dim || P_mat.n_cols != dim || J_mat.n_rows != dim || J_mat.n_cols != dim)
    {
        std::cout << "P_mat or J_mat has wrong dimension!";
        return 1;
    }
    double *P_mat_gpu, *J_mat_gpu;
    hipMalloc((void **)&P_mat_gpu, sizeof(double) * P_mat.n_elem);
    hipMalloc((void **)&J_mat_gpu, sizeof(double) * J_mat.n_elem);
    
    if (sort_AO){
        arma::mat P_mat_temp = P_mat(m_molbasis.mAOs_sorted_index, m_molbasis.mAOs_sorted_index);
        hipMemcpy(P_mat_gpu, P_mat_temp.memptr(), sizeof(double) * P_mat.n_elem, hipMemcpyHostToDevice);
    }else
        hipMemcpy(P_mat_gpu, P_mat.memptr(), sizeof(double) * P_mat.n_elem, hipMemcpyHostToDevice);

    // to find the maximum element in Schwarz_mat
    thrust::device_ptr<double> dev_ptr = thrust::device_pointer_cast(Schwarz_mat);
    thrust::device_ptr<double> max_ptr = thrust::max_element(dev_ptr, dev_ptr + dim * dim);
    double schwarz_max = *max_ptr;

    // Evaluate the J matrix
    int ok = 0;
    ok = eval_Jmat_RSCF(m_molbasis_gpu, rys_root, Schwarz_mat, shreshold, schwarz_max, P_mat_gpu, J_mat_gpu, rys_root_dim1);

    // copy J_mat from GPU to CPU
    hipMemcpy(J_mat.memptr(), J_mat_gpu, sizeof(double) * J_mat.n_elem, hipMemcpyDeviceToHost);

    if (sort_AO)
        J_mat = J_mat(m_molbasis.mAOs_sorted_index_inv, m_molbasis.mAOs_sorted_index_inv);

    return ok;
}
int HartreeFock_Rys_gpu::eval_K(arma::mat &P_mat, arma::mat &K_mat)
{

    size_t dim = m_molbasis.mAOs.size();
    // check the size of P_mat and K_mat
    if (P_mat.n_rows != dim || P_mat.n_cols != dim || K_mat.n_rows != dim || K_mat.n_cols != dim)
    {
        std::cout << "P_mat or K_mat has wrong dimension!";
        return 1;
    }
    double *P_mat_gpu, *K_mat_gpu;
    hipMalloc((void **)&P_mat_gpu, sizeof(double) * P_mat.n_elem);
    hipMalloc((void **)&K_mat_gpu, sizeof(double) * K_mat.n_elem);
    
    if (sort_AO){
        arma::mat P_mat_temp = P_mat(m_molbasis.mAOs_sorted_index, m_molbasis.mAOs_sorted_index);
        hipMemcpy(P_mat_gpu, P_mat_temp.memptr(), sizeof(double) * P_mat.n_elem, hipMemcpyHostToDevice);
    }else
        hipMemcpy(P_mat_gpu, P_mat.memptr(), sizeof(double) * P_mat.n_elem, hipMemcpyHostToDevice);

    // to find the maximum element in Schwarz_mat
    thrust::device_ptr<double> dev_ptr = thrust::device_pointer_cast(Schwarz_mat);
    thrust::device_ptr<double> max_ptr = thrust::max_element(dev_ptr, dev_ptr + dim * dim);
    double schwarz_max = *max_ptr;

    // evaluate the K matrix (two-electron part)
    int ok = 0;
    ok = eval_Kmat_RSCF(m_molbasis_gpu, rys_root, Schwarz_mat, shreshold, schwarz_max, P_mat_gpu, K_mat_gpu, rys_root_dim1);

    // copy K_mat from GPU to CPU
    hipMemcpy(K_mat.memptr(), K_mat_gpu, sizeof(double) * K_mat.n_elem, hipMemcpyDeviceToHost);

    if (sort_AO)
        K_mat = K_mat(m_molbasis.mAOs_sorted_index_inv, m_molbasis.mAOs_sorted_index_inv);

    return ok;
}